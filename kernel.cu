#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernel.h"

void matrixMult(float* matrix_M, float* matrix_N, float* matrix_P, int width, float *execTime) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int size = width*width * sizeof(float);

    float* M_copy, *N_copy, *P_copy;

    hipMalloc((void**)&M_copy, size);
    hipMalloc((void**)&N_copy, size);
    hipMalloc((void**)&P_copy, size);

    hipMemcpy(M_copy, matrix_M, size, hipMemcpyHostToDevice);
    hipMemcpy(N_copy, matrix_N, size, hipMemcpyHostToDevice);    
    
    dim3 blockDimension(16,16, 1);
    dim3 gridDimension;

    gridDimension.x = (width + blockDimension.x - 1) / blockDimension.x;
    gridDimension.y = (width + blockDimension.y - 1) / blockDimension.y;
    gridDimension.z = 1;

    hipEventRecord(start, 0);


    //kernel invocation here
    matrixMultKernel<<<gridDimension, blockDimension>>>(M_copy, N_copy, P_copy, width);


    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(execTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(matrix_P, P_copy, size, hipMemcpyDeviceToHost);

    hipFree(M_copy);
    hipFree(N_copy);
    hipFree(P_copy);

}

__global__ void matrixMultKernel(float* M_copy, float* N_copy, float* P_copy, int width) {

    int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
    int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

    float dotProduct = 0;


    if (rowIdx < width && colIdx < width) {
     
        for (int k = 0; k < width; k++) {
            dotProduct += M_copy[rowIdx*width + k] * N_copy[k*width + colIdx];
        }

        P_copy[rowIdx*width + colIdx] = dotProduct;

    }

}